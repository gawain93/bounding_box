#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "csgpu.h"
#include <iostream>
#include "stdio.h"

__global__ void add(int a, int b, int *c)
{
    *c = a + b;
}
__device__  unsigned int  my_rand(unsigned int *seed) {

    // constants for random no gen.
    unsigned long a = 16807;
    unsigned long m = 2147483647;   	// 2^31 - 1
    unsigned long x = (unsigned long) *seed;

    x = (a * x)%m;

    *seed = (unsigned int) x;

    return x;
}



__global__ void run(int xNr, int yNr, int zNr,int partNr,                 // inputs: number of grids in x,y and z , max and min length in 3 dimensitions
                    float3 minPt, float3 maxPt,float gridsize,
                    float3 *cloudpos,  float3 *cloudhsv, int cloudSize,
                    float3 *partpos,  float3 *partrot, float *d_refhist, float *d_weights, float modelsize)
{

//    int id = blockIdx.x* blockDim.x + threadIdx.x;
//    printf("ind of %i is started\n", id);

    int id =  blockIdx.x;
    int pId = threadIdx.x;
    int nrPerThr = (int)ceil( ((float)cloudSize) / ((float)blockDim.x) );
//    int id =  threadIdx.x;
//    int pId = blockIdx.x;
//    int nrPerThr = 1;//(int)ceil( ((float)cloudSize) / ((float)threadDim.x) );

//    int id =  blockIdx.x*blockDim.x + threadIdx.x;
//    int pId = threadIdx.x;

    if (id+1>partNr)  {return;}
    if (pId+1>cloudSize) {return;}
    float px,py,pz,roll,pitch,yaw;

    px = partpos[id].x; py = partpos[id].y; pz = partpos[id].z;
    roll = partrot[id].x; pitch = partrot[id].y; yaw = partrot[id].z;

    float t[3][3];

    float A = cosf (yaw), B = sinf (yaw), C = cosf (pitch), D = sinf (pitch),
    E = cosf (roll), F = sinf (roll), DE = D*E, DF = D*F;
    t[0][0] = A*C; t[0][1] = A*DF - B*E; t[0][2] = B*F + A*DE;// t[0][3] = px;
    t[1][0] = B*C; t[1][1] = A*E + B*DF; t[1][2] = B*DE - A*F;// t[1][3] = py;
    t[2][0] = -D;  t[2][1] = C*F;        t[2][2] = C*E;       // t[2][3] = pz;
//    t[3][0] = 0.f; t[3][1] = 0.f;        t[3][2] = 0.f;        t[3][3] = 1.f;


    //printf("time of ind %i is %i\n", id, clock());
//    int indOffset = histsize * id; int nrPerThr;
//     __shared__ float hists[10000]; indOffset=0;
//    nrPerThr = (int)ceil((float)histsize/blockDim.x);
//    for(int i=pId*nrPerThr; i<min(nrPerThr*(pId+1),histsize); i++){
//        hists[i]=0.f;
//    }
//    __syncthreads();



//    printf("number per thread is %i\n", nrPerThr);

    //    {printf("threadidx is %i start is  %i end is %i\n",pId, nrPerThr*pId, min(nrPerThr*(pId+1),cloudSize));}




    float thisCorr = 0.f;

//    int thres = (int)ceil(512.f/(float)partNr * 100.f);
//    unsigned int seed = blockIdx.x+threadIdx.x;
    for(int i=nrPerThr*pId; i<min(nrPerThr*(pId+1),cloudSize); i++){
//    for(int i=0; i<cloudSize; i++){

//        unsigned int rand_ = my_rand(&seed)%100;
//        if (rand_>thres){continue;}
//        if ((i+id+pId)% 5 !=0){continue;}

        float _x = cloudpos[i].x, _y = cloudpos[i].y, _z = cloudpos[i].z;

        float x = t[0][0]*_x + t[0][1]*_y + t[0][2]*_z + px;
        float y = t[1][0]*_x + t[1][1]*_y + t[1][2]*_z + py;
        float z = t[2][0]*_x + t[2][1]*_y + t[2][2]*_z + pz;
        float h=cloudhsv[i].x, s=cloudhsv[i].y, v=cloudhsv[i].z;


        int grayindex;
        float weightgray, weightcolor;
        int hindex1,hindex2;
        float hweight1, hweight2;
        h = h*360.0f;
//        if(h>=0   && h <=60)  {hindex1=0; hindex2=1; hweight2= h/60.0f;          hweight1=1.0f -hweight2;}
//        if(h>60  && h <=120)  {hindex1=1; hindex2=2; hweight2= (h-60.0f)/60.0f;  hweight1=1.0f -hweight2;}
//        if(h>120 && h <=180)  {hindex1=2; hindex2=3; hweight2= (h-120.0f)/60.0f; hweight1=1.0f -hweight2;}
//        if(h>180 && h <=240)  {hindex1=3; hindex2=4; hweight2= (h-180.0f)/60.0f; hweight1=1.0f -hweight2;}
//        if(h>240 && h <=300)  {hindex1=4; hindex2=5; hweight2= (h-240.0f)/60.0f; hweight1=1.0f -hweight2;}
//        if(h>300 && h <=360)  {hindex1=5; hindex2=0; hweight2= (h-300.0f)/60.0f; hweight1=1.0f -hweight2;}

        hindex1 = (int)floor(h/60.f);
        hindex2=((hindex1+1)%6);


        hweight2 = (h- 60.f*(float)hindex1)/60.f; hweight1 = 1.f - hweight2;

        if(v<0.5) {grayindex = 6;}
        else{grayindex = 7;}
        if( v<0.2f || s<0.1f){
            weightcolor=0.0f; weightgray=1.0f;
        }else{
            weightcolor = pow(s,(0.14f * pow(1.0f/v,0.9f)));
            weightgray  = 1.f - weightcolor;
        }
//        weightcolor=0.f; weightgray=1.f; grayindex=7;

        hweight1 = weightcolor * hweight1;
        hweight2 = weightcolor * hweight2;

        int xInd, yInd, zInd, theInd;
        xInd = (int)floorf( (x - minPt.x)/gridsize );
        if((xInd >= xNr) || (xInd<0)) {continue;}
        yInd = (int)floorf( (y - minPt.y)/gridsize );
        if((yInd >= yNr) || (yInd<0)) {continue;}
        zInd = (int)floorf( (z - minPt.z)/gridsize );
        if((zInd >= zNr) || (zInd<0)) {continue;}
        theInd = zInd*xNr*yNr + xInd*yNr + yInd;

//        printf("indexs are %i %i %i\n", xInd, yInd, zInd);
//        x = x - gridsize*(float)xInd - minPt.x;
//        y = y - gridsize*(float)yInd - minPt.y;
//        z = z - gridsize*(float)zInd - minPt.z;
//        int xPlusInd = zInd*xNr*yNr + (xInd+1)*yNr + yInd;
//        int yPlusInd = zInd*xNr*yNr + xInd*yNr + (yInd+1);
//        int zPlusInd = (zInd+1)*xNr*yNr + xInd*yNr + yInd;
//        float weight1, weight2;
//        float incrValue =1.f;

//        weight2 = x/gridsize; weight1 = 1.f-weight2;  //weight2 *= weight2; weight1*=weight1;
//        thisCorr += (refhist[hindex1 + 24*theInd + 0] *  hweight1*weight1*incrValue);
//        thisCorr += (refhist[hindex1 + 24*xPlusInd + 0] *   hweight1*weight2*incrValue);
//        thisCorr += (refhist[hindex2 + 24*theInd + 0]   *   hweight2*weight1*incrValue);
//        thisCorr += (refhist[hindex2 + 24*xPlusInd + 0]  *   hweight2*weight2*incrValue);
//        thisCorr += (refhist[grayindex + 24*theInd + 0]  *   weightgray*weight1*incrValue);
//        thisCorr += (refhist[grayindex + 24*xPlusInd + 0] *   weightgray*weight2*incrValue);

//        weight2 = y/gridsize; weight1 = 1.f-weight2;  //weight2 *= weight2; weight1*=weight1;
//        thisCorr += (refhist[hindex1 + 24*theInd + 8] *   hweight1*weight1*incrValue);
//        thisCorr += (refhist[hindex1 + 24*yPlusInd + 8] *   hweight1*weight2*incrValue);
//        thisCorr += (refhist[hindex2 + 24*theInd + 8]   *   hweight2*weight1*incrValue);
//        thisCorr += (refhist[hindex2 + 24*yPlusInd + 8]  *   hweight2*weight2*incrValue);
//        thisCorr += (refhist[grayindex + 24*theInd + 8]  *   weightgray*weight1*incrValue);
//        thisCorr += (refhist[grayindex + 24*yPlusInd + 8] *   weightgray*weight2*incrValue);

//        weight2 = z/gridsize; weight1 = 1.f-weight2; // weight2 *= weight2; weight1*=weight1;
//        thisCorr += (refhist[hindex1 + 24*theInd + 16 ] *  hweight1*weight1*incrValue);
//        thisCorr += (refhist[hindex1 + 24*zPlusInd + 16 ] *   hweight1*weight2*incrValue);
//        thisCorr += (refhist[hindex2 + 24*theInd + 16 ]  *   hweight2*weight1*incrValue);
//        thisCorr += (refhist[hindex2 + 24*zPlusInd + 16 ]  *   hweight2*weight2*incrValue);
//        thisCorr += (refhist[grayindex + 24*theInd + 16 ]  *   weightgray*weight1*incrValue);
//        thisCorr += (refhist[grayindex + 24*zPlusInd + 16 ] *   weightgray*weight2*incrValue);


        x = x - gridsize*(float)xInd - minPt.x;
        y = y - gridsize*(float)yInd - minPt.y;
        z = z - gridsize*(float)zInd - minPt.z;

//        float numsafe=1000.f;
//        x *= numsafe; y*=numsafe; z*=numsafe; gridsize*=numsafe ;

        float x_=gridsize-x,  y_=gridsize-y,  z_=gridsize-z;
//        int xPInd = zInd*xNr*yNr + (xInd+1)*yNr + yInd; //theInd+yNr;
//        int xyPInd = zInd*xNr*yNr + (xInd+1)*yNr + yInd+1; //xPInd+1;
//        int xzPInd = (zInd+1)*xNr*yNr + (xInd+1)*yNr + yInd; //theInd+xNr*yNr+yNr;
//        int xyzPInd = (zInd+1)*xNr*yNr + (xInd+1)*yNr + yInd+1; //xzPInd+1;
//        int yPInd = zInd*xNr*yNr + xInd*yNr + (yInd+1);// theInd+1;
//        int yzPInd = (zInd+1)*xNr*yNr + xInd*yNr + (yInd+1); //yPInd+xNr*yNr;
//        int zPInd = (zInd+1)*xNr*yNr + xInd*yNr + yInd; //theInd + xNr*yNr;
        int xPInd   = theInd + yNr;
        int xyPInd  = xPInd +1;
        int xzPInd  = theInd + xNr*yNr+yNr;
        int xyzPInd = xzPInd +1;
        int yPInd   = theInd +1;
        int yzPInd  = yPInd+xNr*yNr;
        int zPInd   = theInd + xNr*yNr;


        float weight; int offset;
        float gridsize3= 0.000001f;//gridsize*gridsize*gridsize;

//        weight = max(0.f, (bandwidth-dist0))/gridsize;
        weight = max(0.f, ((x_*y_*z_)/gridsize3));
        offset = 8*theInd;
        thisCorr += (d_refhist[hindex1 + offset] * hweight1*weight);
        thisCorr += (d_refhist[hindex2 + offset] * hweight2*weight);
        thisCorr += (d_refhist[grayindex + offset] * weightgray*weight);

//        weight = max(0.f, (bandwidth-distxP))/gridsize;
        weight = max(0.f, ((x*y_*z_)/gridsize3));
        offset = 8*xPInd;
        thisCorr += (d_refhist[hindex1 + offset] * hweight1*weight);
        thisCorr += (d_refhist[hindex2 + offset] * hweight2*weight);
        thisCorr += (d_refhist[grayindex + offset] * weightgray*weight);

//        weight = max(0.f, (bandwidth-distxyP))/gridsize;
        weight = max(0.f, ((x*y*z_)/gridsize3));
        offset = 8*xyPInd;
        thisCorr += (d_refhist[hindex1 + offset] * hweight1*weight);
        thisCorr += (d_refhist[hindex2 + offset] * hweight2*weight);
        thisCorr += (d_refhist[grayindex + offset] * weightgray*weight);

//        weight = max(0.f, (bandwidth-distxzP))/gridsize;
        weight = max(0.f, ((x*y_*z)/gridsize3));
        offset = 8*xzPInd;
        thisCorr += (d_refhist[hindex1 + offset] * hweight1*weight);
        thisCorr += (d_refhist[hindex2 + offset] * hweight2*weight);
        thisCorr += (d_refhist[grayindex + offset] * weightgray*weight);

//        weight = max(0.f, (bandwidth-distxyzP))/gridsize;
        weight = max(0.f, ((x*y*z)/gridsize3));
        offset = 8*xyzPInd;
        thisCorr += (d_refhist[hindex1 + offset] * hweight1*weight);
        thisCorr += (d_refhist[hindex2 + offset] * hweight2*weight);
        thisCorr += (d_refhist[grayindex + offset] * weightgray*weight);

//        weight = max(0.f, (bandwidth-distyP))/gridsize;
        weight = max(0.f, ((x_*y*z_)/gridsize3));
        offset = 8*yPInd;
        thisCorr += (d_refhist[hindex1 + offset] * hweight1*weight);
        thisCorr += (d_refhist[hindex2 + offset] * hweight2*weight);
        thisCorr += (d_refhist[grayindex + offset] * weightgray*weight);

//        weight = max(0.f, (bandwidth-distyzP))/gridsize;
        weight = max(0.f, ((x_*y*z)/gridsize3));
        offset = 8*yzPInd;
        thisCorr += (d_refhist[hindex1 + offset] * hweight1*weight);
        thisCorr += (d_refhist[hindex2 + offset] * hweight2*weight);
        thisCorr += (d_refhist[grayindex + offset] * weightgray*weight);

//        weight = max(0.f, (bandwidth-distzP))/gridsize;
        weight = max(0.f, ((x_*y_*z)/gridsize3));
        offset = 8*zPInd;
        thisCorr += (d_refhist[hindex1 + offset] * hweight1*weight);
        thisCorr += (d_refhist[hindex2 + offset] * hweight2*weight);
        thisCorr += (d_refhist[grayindex + offset] * weightgray*weight);
    }

    atomicAdd(&d_weights[id], thisCorr);
}

std::vector<float> CSGPU::compute()
{

    hipEvent_t evt1, evt2;
    float t;
    hipEventCreate(&evt1);
    hipEventCreate(&evt2);
    hipEventRecord(evt1, 0);


//    float *d_weights;


//    hipMalloc((void **) &d_weights, partNr*sizeof(float));
    hipMemset(d_weights, 0.f, partNr*sizeof(float));               //  fill the memory for d_weights


//    float3 *d_cloudpos, *d_cloudhsv, *d_partpos, *d_partrot;
//    float *d_refhist;
    hipMalloc((void **) &d_cloudpos, cloudSize*sizeof(float3));
    hipMalloc((void **) &d_cloudhsv, cloudSize*sizeof(float3));
//    hipMalloc((void **) &d_partpos, partNr*sizeof(float3));
//    hipMalloc((void **) &d_partrot, partNr*sizeof(float3));

    hipMemcpy(d_cloudpos, cloudpos, cloudSize*sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(d_cloudhsv, cloudhsv, cloudSize*sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(d_partpos, partpos, partNr*sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(d_partrot, partrot, partNr*sizeof(float3), hipMemcpyHostToDevice);
    hipEventRecord(evt2, 0);
    hipEventSynchronize(evt2);
    hipEventElapsedTime(&t, evt1, evt2);
    printf ("Time for transfer datat to GPU: %f ms\n", t);




    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    // Do calculation on device:

    int threadDim = std::min( (int)cloudSize,  1024);
//    dim3 block(20,20);

     run<<<partNr, threadDim>>>(xNr, yNr, zNr, partNr,             // each particle is a block and the number of cloud stands for threads
                     minPt, maxPt, gridsize,
                     d_cloudpos,  d_cloudhsv, cloudSize,
                          d_partpos, d_partrot, d_refhist, d_weights, float(modelsize));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf ("Time for the kernel: %f ms\n", time);

    hipMemcpy(weights, d_weights, partNr*sizeof(float), hipMemcpyDeviceToHost);

    //hipFree(d_weights);
    hipFree(d_cloudpos);   hipFree(d_cloudhsv);
    //hipFree(d_partpos); hipFree(d_partrot);
    std::vector<float> out; out.resize(partNr);
    for (size_t i=0; i<partNr; i++){
        out[i] = weights[i];
//        std::cout<<out[i]<<",";
    }

    free(cloudpos); free(cloudhsv);
   // free(partpos); free(partrot);

    return out;
}


void CSGPU::loadData(float gridsize_, int xnr_, int ynr_, int znr_, int partnr_)
{
    gridsize=gridsize_; xNr=xnr_; yNr=ynr_; zNr=znr_; partNr=partnr_; //cloudSize=cloudsize_;
    partpos = (float3 *)malloc(partNr * sizeof(float3));
    partrot = (float3 *)malloc(partNr * sizeof(float3));
    histsize =(xNr+1) * (yNr+1) * (zNr+1) * 8;// 24*refcloudSize;//
    refhist = (float *)malloc(histsize * sizeof(float));
}

void CSGPU::uploadRefHist()
{
    hipMalloc((void **) &d_refhist, histsize*sizeof(float));
    hipMemcpy(d_refhist, refhist, histsize*sizeof(float), hipMemcpyHostToDevice);
//    free(refhist);
    hipMalloc((void **) &d_partpos, partNr*sizeof(float3));
    hipMalloc((void **) &d_partrot, partNr*sizeof(float3));
    hipMalloc((void **) &d_weights, partNr*sizeof(float));
    weights = (float *)malloc(partNr * sizeof(float));
}

void CSGPU::uploadCurrentCloud(int cloudSize_)
{
    cloudSize = cloudSize_;
    cloudpos = (float3 *)malloc(cloudSize * sizeof(float3));  //Allocates a block of size bytes of memory, returning a pointer to the beginning of the block.
    cloudhsv = (float3 *)malloc(cloudSize * sizeof(float3));
}

void CSGPU::reloadData(float gridsize_, int xnr_, int ynr_, int znr_, int partnr_)
{
    free(refhist);
    gridsize=gridsize_; xNr=xnr_; yNr=ynr_; zNr=znr_; partNr=partnr_; //cloudSize=cloudsize_;
    histsize =(xNr+1) * (yNr+1) * (zNr+1) * 8;// 24*refcloudSize;//
    refhist = (float *)malloc(histsize * sizeof(float));
}

void CSGPU::reuploadRefHist()
{
    hipFree(d_refhist);
    hipMalloc((void **) &d_refhist, histsize*sizeof(float));
    hipMemcpy(d_refhist, refhist, histsize*sizeof(float), hipMemcpyHostToDevice);
}
